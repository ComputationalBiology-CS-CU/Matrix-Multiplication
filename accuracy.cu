#include "hip/hip_runtime.h"
/*
    Reference: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.
        html#ixzz4CtH09yed 
*/

#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <iostream>
#include <cmath>
using namespace std;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
    int width;
    int height;
    float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 20

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; 
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipError_t err = hipMalloc(&d_A.elements, size);
    cout << "CUDA malloc A: " << hipGetErrorString(err) << endl;
    err = hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    cout << "Copy A to device: " << hipGetErrorString(err) << "\n" << endl;

    Matrix d_B;
    d_B.width = B.width; 
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    err = hipMalloc(&d_B.elements, size);
    cout << "CUDA malloc B: " << hipGetErrorString(err) << endl;
    err = hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    cout << "Copy B to device: " << hipGetErrorString(err) << "\n" << endl;

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; 
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    err = hipMalloc(&d_C.elements, size);
    cout << "CUDA malloc C: " << hipGetErrorString(err) << endl;

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((B.width + dimBlock.x - 1) / dimBlock.x, 
        (A.height + dimBlock.y - 1) / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    err = hipDeviceSynchronize();
    cout << "Run kernel: " << hipGetErrorString(err) << endl;

    // Read C from device memory
    err = hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    cout << "Copy C off of device: " << hipGetErrorString(err) << "\n" << endl;

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0.0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Boundary check
    if (row < A.height && col < B.width) {
        for (int e = 0; e < A.width; ++e)
        Cvalue += (A.elements[row * A.width + e]) * 
                (B.elements[e * B.width + col]);
    }

    C.elements[row * C.width + col] = Cvalue;
}

int main(int argc, char const *argv[])
{
    Matrix A, B, C, D;
    int a1, a2, b1, b2;
    int i, j, k;
    float sum = 0.0, param = 0.0, square = 0.0;

    srand(time(NULL));

    // Get dimensions of A and B
    // Run $ ./matrixMul 1 1000000 400
    a1 = atoi(argv[1]); // A's height
    a2 = atoi(argv[2]); // A's width
    b1 = a2; // B's height
    b2 = atoi(argv[3]); // B's width

    A.height = a1;
    A.width = a2;
    A.elements = new float[A.width * A.height];

    B.height = b1;
    B.width = b2;
    B.elements = new float[B.width * B. height];

    C.height = A.height;
    C.width = B.width;
    C.elements = new float[C.width * C.height];

    D.height = A.height;
    D.width = B.width;
    D.elements = new float[D.width * D.height];

    // Fill A and B with random floats
    for (i = 0; i < A.height; ++i) 
        for (j = 0; j < A.width; ++j) 
            A.elements[i * A.width + j] = (float)(rand() % 100);
            //A.elements[i * A.width + j] = ((float)rand() / (float)RAND_MAX) * 100;

    for (i = 0; i < B.height; ++i) 
        for (j = 0; j < B.width; ++j) 
            B.elements[i * B.width + j] = (float)(rand() % 100);
            //B.elements[i * B.width + j] = ((float)rand() / (float)RAND_MAX) * 100;
    /*
    for (i = 0; i < D.height; ++i) 
        for (j = 0; j < D.width; ++j) 
            D.elements[i * D.width + j] = ((float)rand() / (float)RAND_MAX) * 100;
    */

    // Vanilla C++ matrix multiplication
    for (i = 0; i < A.height; ++i)
        for (j = 0; j < B.width; ++j)
            for (k = 0; k < A.width; ++k) {
                C.elements[i * C.width + j] += A.elements[i * A.width + k] * 
                    B.elements[k * B.width + j];
            }

    // Call MatMul(), and therefore MatMulKernel()
    MatMul(A, B, D);

    // Compare matrices C and D -- they should be almost identical
    for (i = 0; i < C.height; ++i) {
        for (j = 0; j < C.width; ++j) {
            param = C.elements[i * C.width + j] - D.elements[i * D.width + j];            
            
            //if (param < 0)
                //param = fabsf(param);

            square = pow(param, 2);
            sum += square;
            
            int k = 0;
            if (param > 0 && k < 10) {
                cout << "param is " << param << "; ";
                cout << "square is " << square << "; ";
                cout << "sum is " << sum << endl;
                ++k;
            }
        }
    }
    cout << "Accuracy is: ";
    cout << fixed << sum << endl;

    // Print matrices A, B, C, and D
    /*
    for (i = 0; i < min(10, A.height); ++i) {
        for (j = 0; j < min(10, A.width); ++j)
            cout << fixed << A.elements[i * A.width + j] << "\t";
        
        cout << endl;
    }
    cout << endl;
    
    for (i = 0; i < min(10, B.height); ++i) {
        for (j = 0; j < min(10, B.width); ++j)
            cout << fixed << B.elements[i * B.width + j] << "\t";

        cout << endl;
    }
    cout << endl;

    for (int i = 0; i < min(10, C.height); ++i) {
        for (int j = 0; j < min(10, C.width); ++j) {
            cout << fixed << C.elements[i * C.width + j] << "\t";
        }
        cout << endl;
    }
    cout << endl;

    for (int i = 0; i < min(10, D.height); ++i) {
        for (int j = 0; j < min(10, D.width); ++j) {
            cout << fixed << D.elements[i * D.width + j] << "\t";
        }
        cout << endl;
    }
    cout << endl;
    */
    
    delete[] A.elements;
    delete[] B.elements;
    delete[] C.elements;
    delete[] D.elements;
    
    return 0;
}