#include "hip/hip_runtime.h"
/*
  Reference: http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.
    html#ixzz4CtH09yed 
*/

#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <iostream>
using namespace std;

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride; 
    float* elements;
} Matrix;

// Thread block size
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
#define BLOCK_SIZE 10

__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, Cd.elements, size,hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    float Cvalue = 0;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e)
        Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
    C.elements[row * C.width + col] = Cvalue;
}

int main(int argc, char const *argv[])
{
    clock_t t;
    Matrix A, B, C;
    int a1, a2, b1, b2;

    srand(time(NULL));

    // Get dimensions of A and B
    // Run $ ./matrixMul 1 1000000 400
    a1 = atoi(argv[1]); // A's height
    a2 = atoi(argv[2]); // A's width
    b1 = a2; // B's height
    b2 = atoi(argv[3]); // B's width

    A.height = a1;
    A.width = a2;
    A.elements = new float[A.width * A.height];

    B.height = b1;
    B.width = b2;
    B.elements = new float[B.width * B. height];

    C.height = A.height;
    C.width = B.width;
    C.elements = new float[C.width * C.height];

    // Fill A and B with random floats
    for (int i = 0; i < A.height; ++i)
        for (int j = 0; j < A.width; ++j)
            A.elements[i * A.width + j] = float(rand() % 100);

    for (int i = 0; i < B.height; ++i)
        for (int j = 0; j < B.width; ++j)
            B.elements[i * B.width + j] = float(rand() % 100);

    // Call MatMul(), and therefore MatMulKernel()
    t = clock();

    MatMul(A, B, C);

    // Print time the multiplication took
    t = clock() - t;
    cout << "It took me " << fixed << ((float)t)/CLOCKS_PER_SEC;
    cout << " seconds." << endl;

    // Print C
    for (int i = 0; i < min(10, C.height); ++i) {
        for (int j = 0; j < min(10, C.width); ++j)
            cout << fixed << C.elements[i * C.width + j] << "\t";

        cout << endl;
    }
    cout << endl;

    delete [] A.elements;
    delete [] B.elements;
    delete [] C.elements;
    
    return 0;
}